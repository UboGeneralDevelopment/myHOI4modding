#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"

#include<stdio.h>
#include<string>
#include<fstream>
#include<math.h>

#define STEP 1
#define PI 3.1415926535897932384626433832795f //�~����
#define Filedivide 1�@//�ǂݍ��݃t�@�C���̕����B���ǎg��Ȃ������B
#define ThreadsPerBlock 512 //���݂� CUDA �ł� 512 �� MAX
#define angle_div 50 //�]�����̕�����

/*
���j

�Ƃ肠�����A�c�Ɖ��̕����������肵�ĕ����������摜�𐶐�������B���������͕��������Ƃ͈ˑ�����K�v���͂Ȃ����A�Ƃ肠�����͌��������͕��������ƈ�v������B
�܂��A����Ă���͓̂��ߗ��̍��v�ł͂Ȃ��A���̍ő�l�݂̂𗘗p�B��蓧�ߑ�"�炵��"�Ȃ�

�Ƃ肠�����A�摜�ꖇ�ɂ��P�x�����v�����]���l���Z�肵�A������e�p���ɑ΂��ĉ摜�]���l���Ƃɓ��ς��Ƃ��č��v�������̂����̎p���̍ŏI�]���l�Ƃ���B
�摜�ꖇ�̕]���l�́A�P�x�l�̍��v�Ƃ���B

�t�@�C���ǂݍ��ݏI�����_�ŁA���e�p�x�����Ǝ��s�p�x�񐔐ݒ���s���B
���e�L�����o�X�쐬�����C�}�[�`���O�v�Z���]���l�Z�o���������J���@�܂ł���A�̃v���Z�X�Ƃ��ă��W���[�����B
���e�o���t�@�C���Ɠ��e���t�@�C����main���ō쐬���āA��]�ݒ肾���O���֐��ɓ�����B

����A���������̐��Y���ł��Ȃ��Bfucking shit


�t���O�����g������̂ŁAmalloc�Ƃ�free��for���̊O�B

*/

__global__ void forward_marching_GPU(unsigned short *d_input_volume, float *d_proj1, float *d_ray_position, float step_x, float step_y, float step_z, Params params) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	int j;
	float v = 0;//�P�x�̊i�[��
	d_proj1[idx] = 0;
	//���C���ɂ��ēƗ��v�Z���Ă��邽�߁Aid�͈�ł����B
		for (j = 0; j < params.projection_radius / 2.0 / STEP; j++) {//���v���X�e�b�v�𓥂ނ���j�ŕ\���Ă���
			int x = (int)(d_ray_position[idx * 3] + j*step_x);//�����̈�X�e�b�v���ƂɁAxyz�����ɉ��{�N�Z�����i�ނ����v�Z���Aint�Ŋۂ߂��čŏI�I�ȓ_���W�𓾂Ă���B
			int y = (int)(d_ray_position[idx * 3 + 1] + j*step_y);
			int z = (int)(d_ray_position[idx * 3 + 2] + j*step_z);
			if (x > 0 && y > 0 && z > 0 && x < params.voxels_x && y < params.voxels_y && z < params.voxels_z) {
				//������̓_���A�{�����[���̒u���Ă���{�b�N�X��Ԃɂ��鎞�A
				//v += d_input_volume[x + y * params.voxels_x + z * params.voxels_x * params.voxels_y];//���ɂ��̍��\��ɂ���g���O���t�̃{�N�Z���̒l�����Z���Ă����B
				if (v < d_input_volume[x + y * params.voxels_x + z * params.voxels_x * params.voxels_y]) {//�ő�l���Ƃ�B
					v = d_input_volume[x + y * params.voxels_x + z * params.voxels_x * params.voxels_y];
				}
			}
			int x2 = (int)(d_ray_position[idx * 3] - j*step_x);//�t�����ɂ��}�[�`���O����B
			int y2 = (int)(d_ray_position[idx * 3 + 1] - j*step_y);
			int z2 = (int)(d_ray_position[idx * 3 + 2] - j*step_z);
			if (x2 > 0 && y2 > 0 && z2 > 0 && x2 < params.voxels_x && y2 < params.voxels_y && z2 < params.voxels_z) {
				//������̓_���A�{�����[���̒u���Ă���{�b�N�X��Ԃɂ��鎞�A
				//v += d_input_volume[x2 + y2 * params.voxels_x + z2 * params.voxels_x * params.voxels_y];//���ɂ��̍��\�ɂ���g���O���t�̃{�N�Z���̒l�����Z���Ă����B
				if (v < d_input_volume[x2 + y2 * params.voxels_x + z2 * params.voxels_x * params.voxels_y]) {//�ő�l���Ƃ�B
					v = d_input_volume[x2 + y2 * params.voxels_x + z2 * params.voxels_x * params.voxels_y];
				}
			}
		}
		d_proj1[idx] = v;
}


void ray_start_setting(Params params, float *ray_position) {
	int i, j;
	//�������B���W���S��ɂ����ă��C�̃X�^�[�g�ʒu��ݒ�Bi���L�����o�X��ł�y���W,j���L�����o�X��ł�x���W�ɑΉ�
	for (i = 0; i < params.projection_sides; i++) {
		for (j = 0; j < params.projection_sides; j++) {
			ray_position[(i*params.projection_sides + j) * 3] = j - params.projection_radius / 2;
			ray_position[(i*params.projection_sides + j) * 3 + 1] = i - params.projection_radius / 2;
			ray_position[(i*params.projection_sides + j) * 3 + 2] = 0;
		}
	}

	//�X�^�[�g���W��y,z���ɑ΂���theta,phi������]�������̂��A���s�ړ�������
	float a, b, c;
	for (i = 0; i < params.projection_sides*params.projection_sides; i++) {
		//�܂�y���ɂ����ĉ�]
		a = ray_position[i * 3];
		b = ray_position[i * 3 + 1];
		c = ray_position[i * 3 + 2];
		ray_position[i * 3] = cos(params.projection_phi)*a + sin(params.projection_phi)*c;
		ray_position[i * 3 + 1] = b;
		ray_position[i * 3 + 2] = -sin(params.projection_phi)*a + cos(params.projection_phi)*c;
	 //����z���ɂ����ĉ�]
		a = ray_position[i * 3];
		b = ray_position[i * 3 + 1];
		c = ray_position[i * 3 + 2];
		ray_position[i * 3] = cos(params.projection_theta)*a - sin(params.projection_theta)*b;
		ray_position[i * 3 + 1] = sin(params.projection_theta)*a + cos(params.projection_theta)*b;
		ray_position[i * 3 + 2] = c;
		//�{�����[���̒��S���W�ɕ��s�ړ�

		ray_position[i * 3] = ray_position[i * 3] + params.voxels_x / 2.0 - 0.5;
		ray_position[i * 3 + 1] = ray_position[i * 3 + 1] + params.voxels_y / 2.0 - 0.5;
		ray_position[i * 3 + 2] = ray_position[i * 3 + 2] + params.voxels_z / 2.0 - 0.5;
		//printf("ray_2_position(%f,%f,%f)\n", ray_position[i * 3], ray_position[i * 3 + 1], ray_position[i * 3 + 1]);
	}

}



int main(int argc, char** argv) {//argc�Ƃ��ɂ͋N�����ɓn���ϐ����͂���B

	int i, j, k, l;//�֗��ɃJ�E���g�ȂǂɎg������
	Params params;//�p�����[�^�\����
	FILE *in, *para;//���̓t�@�C���p�̃t�@�C���e��
						  //���̓t�@�C���Ɠ����ϐ��Ԃ̐����̂����̈ꎞ�I�ȃo�b�t�@

	//////////////////////�p�����[�^�ǂݍ���//////////////////////////

	if ((para = fopen(argv[1], "r")) == NULL) {
		printf("FILE do not exist\n");
		exit(0);
	};
	
	//�ݒ�t�@�C������p�����[�^�ǂݍ��݁B�X�y�[�X���󂯂�Ǝ��ɍs���̂Ńt�@�C�����ɃX�y�[�X�͂��Ȃ��悤�ɁB
	fscanf(para, "%s", params.in_path_name);
	fscanf(para, "%s", params.out_name);
	fscanf(para, "%d", &params.in_offset);
	fscanf(para, "%f %f", &params.source_object_distance, &params.source_detector_distance);//�����g��Ȃ�
	fscanf(para, "%d %d %d", &params.voxels_x, &params.voxels_y, &params.voxels_z);
	fscanf(para, "%f %f", &params.projection_phi, &params.projection_theta);//���܂����Ȃ�
	fclose(para);

	printf("projection angle division number in form of phi theta\n");//���e�p�x�̐ݒ�B�����œǂݍ��ޓ��e�͂悭�ύX����B
	scanf_s("%f %f", &params.projection_phi, &params.projection_theta);

	params.projection_div_phi = 8;//�Ƃ肠������`�B������scan�ł�݂��܂��Ă��悢�B
	params.projection_div_theta = 8;

	params.projection_radius = sqrt(params.voxels_x * params.voxels_x + params.voxels_y *params.voxels_y + params.voxels_z * params.voxels_z);//���e���̈�Ђ̒���
	params.projection_sides = (int)params.projection_radius;//�{�N�Z�������A�܂萮���Ɋۂ߂��ꍇ�B

	printf(" input %s\n output %s\n distance %f %f\n voxels %d %d %d\n angle phi %f theta %f\n angle division phi %d theta %d\n projection radius %f projection sides %d",
		params.in_path_name, params.out_name,
		params.source_object_distance, params.source_detector_distance,
		params.voxels_x, params.voxels_y, params.voxels_z,
		params.projection_phi, params.projection_theta,
		params.projection_div_phi, params.projection_div_theta,
		params.projection_radius,params.projection_sides);//�p�����[�^�����o���B
	//////////////////////�p�����[�^�ǂݍ��ݏI��//////////////////////////

	//////////////////////�ǂݍ��݂Ɗe�탁�����m��//////////////////////////
	if ((in = fopen(params.in_path_name, "rb")) == NULL) {
		printf("FILE do not exist_1\n");
		exit(0);//�R�}���h���C������̎��s�ł̓t���̃p�X���w�肵�Ȃ���Ό�����Ȃ��B������\��2�񂩂��B���邢�͐ݒ�e�L�X�g�𒼐ڃv���O�����ɕ��蓊���Ă��悢�B
	}
	printf("load_success\n");
	
	unsigned short *input_volume = new unsigned short[params.voxels_y*params.voxels_x*params.voxels_z];
	//���ʃT�C�Y�~���݂�1�����z��쐬
	printf("memory_success\n");

	fseek(in, params.in_offset, SEEK_SET);//�ǂݍ��ݎ��ɃI�t�Z�b�g�ړ�
	fread(input_volume, 2, params.voxels_x*params.voxels_y*params.voxels_z, in);//�{�����[���ǂݍ���
	fclose(in);
	printf("loading_success\n");
	
	float *proj1 = new float[params.projection_sides*params.projection_sides];
	//���e���̃������m��
	float *ray_position = new float[params.projection_sides*params.projection_sides * 3];
	//�L�����o�X�T�C�Y��3�{�̓��e���C�̃X�^�[�g�ʒu���i�[���郁�����Bxyz���W�Őݒ�
	float ray_step[3];
	//���C�̊i�[�e��쐬
	char name[1000];
	//�����o���p�̖��O
	float *temp = new float[params.projection_div_phi*params.projection_div_theta*3];
	//�]���l�Ɠ��e�p�x�̃�����

	//////////////////////�ȉ�GPU�ł̃������m��
	float* d_proj1;
	float* d_ray_position;
	unsigned short* d_input_volume;
	//GPU�������m��
	hipMalloc(&d_proj1, sizeof(float)*params.projection_sides*params.projection_sides);
	hipMalloc(&d_ray_position,sizeof(float)*params.projection_sides*params.projection_sides * 3);
	hipMalloc(&d_input_volume, sizeof(unsigned short)*params.voxels_x*params.voxels_y*params.voxels_z);

	/////////////////////�ǂݍ��݂Ɗe�탁�����m�ۏI��//////////////////////////

	//////////////////////�J��Ԃ��Ώ�////////////////////////
	
	for (i = 0; i < params.projection_div_phi; i++) {//�J��Ԃ��񐔂̐ݒ�Ƃ肠�����K��

		//////////////////////���e�������������A���e�X�^�[�g�ʒu�ݒ�A���C�����ݒ�A�쐬//////////////////////////
		printf("projection_setting_start\n");

		for (j = 0; j < params.projection_sides*params.projection_sides; j++) {
			proj1[j] = 0;
		}//���e������������

		ray_start_setting(params, ray_position);
		//�X�^�[�g�ʒu�ݒ�A�X�^�[�g�ʒu�ƁA�p�����[�^�𓊂�����ŉ�]������B

		ray_step[0] = sin(params.projection_phi)*cos(params.projection_theta)*STEP;
		ray_step[1] = sin(params.projection_phi)*sin(params.projection_theta)*STEP;
		ray_step[2] = cos(params.projection_phi)*STEP;
		//���C�̕������p�����[�^�ɂ��������čX�V�B�{�N�Z�����������̂܂܍��W�n�̒����ɑΉ����Ă��邽�߁A���̂܂܃X�e�b�v�Ƃ��ėp�����B
	
		printf("ray,ray_canvas,ray_start_positions are successfully_created\n");
		printf("ray,%f,%f,%f\n",ray_step[0],ray_step[1],ray_step[2]);
		printf("picture pixel %f\n", proj1[params.projection_sides*params.projection_sides / 4]);
		//////////////////////���e�L�����o�X�쐬�I��//////////////////////////


		//////////////////GPU�]������у��C�}�[�`���O�v�Z///////////////////////

		//GPU�Ƀf�[�^�]��
		hipMemcpy(d_proj1, proj1, sizeof(float)*params.projection_sides*params.projection_sides, hipMemcpyHostToDevice);
		hipMemcpy(d_ray_position, ray_position, sizeof(float)*params.projection_sides*params.projection_sides * 3, hipMemcpyHostToDevice);
		hipMemcpy(d_input_volume, input_volume, sizeof(unsigned short)*params.voxels_x*params.voxels_y*params.voxels_z, hipMemcpyHostToDevice);
	
		printf("ray_marching_start\n");
	
		//�u���b�N���ݒ��GPU�v�Z�Ăяo��
		int BlockNum = (params.projection_sides*params.projection_sides + ThreadsPerBlock - 1) / ThreadsPerBlock;
		forward_marching_GPU <<< BlockNum, ThreadsPerBlock >>> ( d_input_volume,  d_proj1, d_ray_position, ray_step[0], ray_step[1], ray_step[2], params);

		//���e���̉��
		hipMemcpy(proj1, d_proj1, sizeof(float)*params.projection_sides*params.projection_sides, hipMemcpyDeviceToHost);
	
		printf("ray_marching_end\n");

		//////////////////////GPU�]�������//////////////////////////


		//////////////////////�]���l�̌v�Z////////////////////
		temp[3*i+1] = params.projection_phi;
		temp[3*i+2] = params.projection_theta;
		for (j = 0; j < params.projection_sides*params.projection_sides; j++) {
			temp[3*i] += proj1[j];
		}
		//////////////////////�]���l�̌v�Z�I��/////////////////////////


		//////////////////////�����o��(�I�v�V����)//////////////////////////
		printf("picture pixel %f\n",proj1[params.projection_sides*params.projection_sides/4]);

		printf("Writing\n");//���������͏����o���B�����o���t�@�C���Ɍ`���̎w��Ȃǂ͂Ȃ��B

		sprintf(name, "%s-float-%dx%d-(%f_%f).raw", params.out_name, params.projection_sides, params.projection_sides,params.projection_phi, params.projection_theta);
		printf("%s", name);
	
		FILE *out;
		out = fopen(name, "wb");
		if (out == NULL) {
			printf("\nFILE cannot open\n");
			exit(0);
		};
	
		//out�ɓ��e�L�����o�X���Ԃ����ށB�Ƃ肠�����̓t���[�g��
		fwrite(proj1, sizeof(float), params.projection_sides*params.projection_sides, out);
	
		fclose(out);
		printf("\nwriting_end\n\n");

	//�R�}���h���C������̎��s�ł͏o�̓t�@�C���̓v���O�����̃t�H���_���ɂł���B���ڃe�L�X�g�𓊂����ނƃe�L�X�g�̂���t�H���_�ɂł���B

		//////////////////////�����o���I��//////////////////////////


		//////////////////////�J��Ԃ��̎��̎p���X�V//////////////////////////
		params.projection_phi = params.projection_phi + PI / params.projection_div_phi;
		//params.projection_theta = params.projection_theta + PI / params.projection_div_theta;
		//////////////////////�p���X�V�I��//////////////////////////

	}
	//////////////////////�J��Ԃ��ΏۏI��////////////////////////


	////////////////�������J��///////////////
	hipFree(d_input_volume);
	hipFree(d_proj1);
	hipFree(d_ray_position);	
	delete[] proj1;
	delete[] ray_position;	
	delete[] input_volume;
	////////////////�������J��///////////////

	//////////////////////�]���l�̔�r����////////////////////
	float vphi, vtheta;
	float *value = new float[angle_div*angle_div];
	float max_value[3];
	for (i = 0; i < params.projection_div_theta*params.projection_div_phi; i++) {
		printf("value %f (phi %f theta %f)\n", temp[3*i], temp[3*i + 1], temp[3*i + 2]);

		for (j = 0; j < angle_div; j++) {
			for (k = 0; k < angle_div; k++) {
				value[j*angle_div + k] = 0;
			}
		}
		for (j = 0; j < angle_div; j++) {
			vphi = PI*2.0f*j/angle_div;
			for (k = 0; k < angle_div; k++) {
				vtheta = PI*2.0f*k / angle_div;
				for (l = 0; l < params.projection_div_phi * params.projection_div_theta ; l++) {
					value[j*angle_div + k] += temp[3*i]*(sin(temp[3 * l + 1])*cos(temp[3 * l + 2])*sin(vphi)*cos(vtheta) + sin(temp[3 * l + 1])*sin(temp[3 * l + 2])*sin(vphi)*sin(vtheta) + cos(temp[3 * l + 1])*cos(vphi));
				}
				//printf("value %f phi %f theta %f\n", value[j*angle_div + k], vphi, vtheta);
				if (max_value[0]<value[j*angle_div + k]) {
					max_value[0] = value[j*angle_div + k];
					max_value[1] = vphi;
					max_value[2] = vtheta;
				}
			}
		}
	}

	printf("max_value %f phi %f theta %f\n", max_value[0], max_value[1], max_value[2]);

	//////////////////////�]���l�̔�r�����I��////////////////
	
	////////////////�������J��///////////////
	delete[] temp;
	delete[] value;
	////////////////�������J��///////////////

	printf("program_end\n");

	return 0;
}