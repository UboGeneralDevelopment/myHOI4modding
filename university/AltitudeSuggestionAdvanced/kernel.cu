#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"

#include<stdio.h>
#include<string>
#include<fstream>
#include<math.h>

#define STEP 1
#define PI 3.1415926535897932384626433832795f //�~����
#define Filedivide 1�@//�ǂݍ��݃t�@�C���̕����B���ǎg��Ȃ������B
#define ThreadsPerBlock 512 //���݂� CUDA �ł� 512 �� MAX
#define angle_div 50 //�]�����̕�����
//#define threshold 10

/*
���j

�c�Ɖ��̕������ɂ��ẮA�ő啪�����ݒ��A�i�ܓx���ƂɎ�����������B���������͕��������Ƃ͕K�������ˑ����Ȃ��B
���_�ɂ�����B����]�͈Ӗ����Ȃ����߁A������x�ɑ΂��Ď��ۂɂ�x+1�������A���̍ŏ��ƍŌ�̒��_�ʒu�ł̎B�����s��Ȃ��B

����Ă���͓̂��ߗ��̍��v�ł͂Ȃ��A���̍ő�l�݂̂𗘗p�B��蓧�ߑ�"�炵��"�Ȃ�B
�ő�l�݂̂��Ƃ�̂ŁA���f����uint16,unsigne int�t�@�C���B���ɓ������̍��v���o��������Ε��ς���B
���e���̏o�͂�����Ȃ��uint16

�����o�����ߗ��Ɗp�x�̃}�b�v��double�ŏo�́B

�t�@�C���ǂݍ��ݏI�����_�ŁA���e�p�x�����Ǝ��s�p�x�񐔐ݒ���s���B
���e�L�����o�X�쐬�����C�}�[�`���O�v�Z���]���l�Z�o���������J���@�܂ł���A�̃v���Z�X�Ƃ��ă��W���[�����B
���e�o���t�@�C���Ɠ��e���t�@�C����main���ō쐬���āA��]�ݒ肾���O���֐��ɓ�����B

�t���O�����g������̂ŁAmalloc�Ƃ�free��for���̊O�B
�{�����[����GPU�ւ̓ǂݍ��݂͈��̂݁B

*/

__global__ void forward_marching_GPU(float *d_input_volume, float *d_proj1, float *d_ray_position, float step_x, float step_y, float step_z, Params params) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	int j = 0;
	
	float x, y, z;
	int xi, yi, zi;
	int before = 0;
	if (params.threshold < d_input_volume[xi + yi * params.voxels_x + zi * params.voxels_x * params.voxels_y]) {
		before = 2;
	}

	float x2, y2, z2;
	int xi2, yi2, zi2;
	int before2 = 0;
	if (params.threshold < d_input_volume[xi + yi * params.voxels_x + zi * params.voxels_x * params.voxels_y]) {
		before2 = 2;
	}

	int on_metal = 0;
	int out_metal = 0;
	float v = 0;//�P�x�̊i�[��
	d_proj1[idx] = 0;


	for (j = 0; j < params.projection_radius / 2.0 / STEP; j++) {//���v���X�e�b�v�𓥂ނ���j�ŕ\���Ă���
		x = d_ray_position[idx * 3] + j*step_x;//�����̈�X�e�b�v���ƂɁAxyz�����ɉ��{�N�Z�����i�ނ����v�Z���Aint�Ŋۂ߂��čŏI�I�ȓ_���W�𓾂Ă���B
		y = d_ray_position[idx * 3 + 1] + j*step_y;
		z = d_ray_position[idx * 3 + 2] + j*step_z;
		xi = x;
		yi = y;
		zi = z;

		if (xi > 0 && yi > 0 && zi > 0 && xi < params.voxels_x && yi < params.voxels_y && zi < params.voxels_z) {

			//v += d_input_volume[xi + yi * params.voxels_x + zi * params.voxels_x * params.voxels_y];
			//������̋P�x�����v
			/*if (v < d_input_volume[xi + yi * params.voxels_x + zi * params.voxels_x * params.voxels_y]) {//�ő�l���Ƃ�B
				v = d_input_volume[xi + yi * params.voxels_x + zi * params.voxels_x * params.voxels_y];
			}*/

			if (params.threshold < d_input_volume[xi + yi * params.voxels_x + zi * params.voxels_x * params.voxels_y]) {
				if (1 > before) {//���^���ɂԂ�����
					++on_metal;
					before = 2;
				}
			}
			else if (params.threshold > d_input_volume[xi + yi * params.voxels_x + zi * params.voxels_x * params.voxels_y]) {
				if (1 < before) {//���^������~�肽
					++out_metal;
					before = 0;
				}
			}

		}
	}

	for (j = 0; j < params.projection_radius / 2.0 / STEP; j++) {//���v���X�e�b�v�𓥂ނ���j�ŕ\���Ă���
		x2 = d_ray_position[idx * 3] - j*step_x;//�����̈�X�e�b�v���ƂɁAxyz�����ɉ��{�N�Z�����i�ނ����v�Z���Aint�Ŋۂ߂��čŏI�I�ȓ_���W�𓾂Ă���B
		y2 = d_ray_position[idx * 3 + 1] - j*step_y;
		z2 = d_ray_position[idx * 3 + 2] - j*step_z;
		xi2 = x2;
		yi2 = y2;
		zi2 = z2;
		if (xi2 > 0 && yi2 > 0 && zi2 > 0 && xi2 < params.voxels_x && yi2 < params.voxels_y && zi2 < params.voxels_z) {

			//v += d_input_volume[xi2 + yi2 * params.voxels_x + zi2 * params.voxels_x * params.voxels_y];
			//������̋P�x�����v
			/*if (v < d_input_volume[xi2 + yi2 * params.voxels_x + zi2 * params.voxels_x * params.voxels_y]) {//�ő�l���Ƃ�B
			v = d_input_volume[xi2 + yi2 * params.voxels_x + zi2 * params.voxels_x * params.voxels_y];
			}*/

			if (params.threshold < d_input_volume[xi2 + yi2 * params.voxels_x + zi2 * params.voxels_x * params.voxels_y]) {
				if (1 > before2) {//���^���ɂԂ�����
					++on_metal;
					before2 = 2;
				}
			}
			else if (params.threshold > d_input_volume[xi2 + yi2 * params.voxels_x + zi2 * params.voxels_x * params.voxels_y]) {
				if (1 < before2) {//���^������~�肽
					++out_metal;
					before2 = 0;
				}
			}

		}
	}

	v = (float)on_metal + (float)out_metal;
	d_proj1[idx] = v;
}


void ray_start_setting(Params params, float *ray_position, float ray_phi, float ray_theta) {
	int i, j;
	//�������B���W���S��ɂ����ă��C�̃X�^�[�g�ʒu��ݒ�Bi���L�����o�X��ł�y���W,j���L�����o�X��ł�x���W�ɑΉ��B
	for (i = 0; i < params.projection_sides; i++) {
		for (j = 0; j < params.projection_sides; j++) {
			ray_position[(i*params.projection_sides + j) * 3] = j - params.projection_radius / 2;
			ray_position[(i*params.projection_sides + j) * 3 + 1] = i - params.projection_radius / 2;
			ray_position[(i*params.projection_sides + j) * 3 + 2] = 0;
		}
	}

	//�X�^�[�g���W��y,z���ɑ΂���theta,phi������]�������̂��A���s�ړ�������
	float a, b, c;
	for (i = 0; i < params.projection_sides*params.projection_sides; i++) {
		//�܂�y���ɂ����ĉ�]
		a = ray_position[i * 3];
		b = ray_position[i * 3 + 1];
		c = ray_position[i * 3 + 2];
		ray_position[i * 3] = cos(ray_phi)*a + sin(ray_phi)*c;
		ray_position[i * 3 + 1] = b;
		ray_position[i * 3 + 2] = -sin(ray_phi)*a + cos(ray_phi)*c;
	 //����z���ɂ����ĉ�]
		a = ray_position[i * 3];
		b = ray_position[i * 3 + 1];
		c = ray_position[i * 3 + 2];
		ray_position[i * 3] = cos(ray_theta)*a - sin(ray_theta)*b;
		ray_position[i * 3 + 1] = sin(ray_theta)*a + cos(ray_theta)*b;
		ray_position[i * 3 + 2] = c;

		//�{�����[���̒��S���W�ɕ��s�ړ�
		ray_position[i * 3] = ray_position[i * 3] + params.voxels_x / 2.0 - 0.5;
		ray_position[i * 3 + 1] = ray_position[i * 3 + 1] + params.voxels_y / 2.0 - 0.5;
		ray_position[i * 3 + 2] = ray_position[i * 3 + 2] + params.voxels_z / 2.0 - 0.5;
		//printf("ray_2_position(%f,%f,%f)\n", ray_position[i * 3], ray_position[i * 3 + 1], ray_position[i * 3 + 1]);
	}

}



int main(int argc, char** argv) {//argc�Ƃ��ɂ͋N�����ɓn���ϐ����͂���B

	int i, j, k, l;//�֗��ɃJ�E���g�ȂǂɎg������
	Params params;//�p�����[�^�\����
	FILE *in, *para;//���̓t�@�C���p�̃t�@�C���e��
						  //���̓t�@�C���Ɠ����ϐ��Ԃ̐����̂����̈ꎞ�I�ȃo�b�t�@

	//////////////////////�p�����[�^�ǂݍ��݂ƃp�����[�^�ݒ�//////////////////////////

	/*
	�ݒ�t�@�C������p�����[�^�ǂݍ��݁B�X�y�[�X���󂯂�Ǝ��ɍs���̂Ńt�@�C�����ɃX�y�[�X�͂��Ȃ��B�ݒ�t�@�C���̏����͎��̒ʂ�
	
	���̓{�����[���t�@�C���̃p�X
	�o�̓t�@�C���̖��O�̈ꕔ
	���̓{�����[���t�@�C���̃I�t�Z�b�g
	)���̓{�����[���̐���-���o�틗��(���p�X�y�[�X���̓{�����[���̐���-��]���S����
	���̓{�����[���̃{�N�Z����X(���p�X�y�[�X)���̓{�����[���̃{�N�Z����Y(���p�X�y�[�X)���̓{�����[���̃{�N�Z����Z
	���e���쐬�̎���phi����������(���p�X�y�[�X)���e���쐬�̎���theta����������

	*/

	if ((para = fopen(argv[1], "r")) == NULL) {
		printf("FILE do not exist\n");
		exit(0);
	};

	fscanf(para, "%s", params.in_path_name);
	fscanf(para, "%s", params.out_name);
	fscanf(para, "%d", &params.in_offset);
	fscanf(para, "%f %f", &params.source_detector_distance, &params.source_object_distance);//�����g��Ȃ�
	fscanf(para, "%d %d %d", &params.voxels_x, &params.voxels_y, &params.voxels_z);
	fscanf(para, "%f %f", &params.projection_div_phi, &params.projection_div_theta);//�����̕������ݒ�B������scanf_s�œǂݍ��݂Ȃ����B
	fclose(para);

	printf("projection angle division number in form of phi theta\n");//���e�p�x�̖����̐ݒ�B�����œǂݍ��ޓ��e�͂悭�ύX����B
	scanf_s("%d %d", &params.projection_div_phi, &params.projection_div_theta);
	printf("threshold\n");//threshold�ݒ�
	scanf_s("%f", &params.threshold);


	params.projection_radius = sqrt(params.voxels_x * params.voxels_x + params.voxels_y *params.voxels_y + params.voxels_z * params.voxels_z);//���e���̈�Ђ̒���
	params.projection_sides = (int)params.projection_radius;//�{�N�Z�������A�܂萮���Ɋۂ߂��ꍇ�B

	printf(" input %s\n output %s\n distance %f %f\n voxels %d %d %d\n angle division phi %d theta %d\n projection radius %f projection sides %d",
		params.in_path_name, params.out_name,
		params.source_object_distance, params.source_detector_distance,
		params.voxels_x, params.voxels_y, params.voxels_z,
		params.projection_div_phi, params.projection_div_theta,
		params.projection_radius, params.projection_sides);//�p�����[�^�����o���B




	//////////////////////�ǂݍ��݂Ɗe�탁�����m��,GPU�]��//////////////////////////
	if ((in = fopen(params.in_path_name, "rb")) == NULL) {
		printf("FILE do not exist_1\n");
		exit(0);//�R�}���h���C������̎��s�ł̓t���̃p�X���w�肵�Ȃ���Ό�����Ȃ��B������\��2�񂩂��B���邢�͐ݒ�e�L�X�g�𒼐ڃv���O�����ɕ��Ă��悢�B
	}
	printf("load_success\n");

	float *input_volume = new float[params.voxels_y*params.voxels_x*params.voxels_z];
	//���ʃT�C�Y�~���݂�1�����z��쐬
	printf("memory_success\n");

	fseek(in, params.in_offset, SEEK_SET);//�ǂݍ��ݎ��ɃI�t�Z�b�g�ړ�
	fread(input_volume, sizeof(float), params.voxels_x*params.voxels_y*params.voxels_z, in);//�{�����[���ǂݍ���
	fclose(in);
	printf("loading_success\n");

	float *proj1 = new float[params.projection_sides*params.projection_sides];
	//���e���̃������m�ہBCPU��
	float *ray_position = new float[params.projection_sides*params.projection_sides * 3];
	//�L�����o�X�T�C�Y��3�{�̓��e���C�̃X�^�[�g�ʒu���i�[���郁�����BCPU���Bxyz���W�Őݒ�
	float ray_step[3];
	//���C�̊i�[�e��쐬
	float ray_phi = 0.0;
	float ray_theta = 0.0;
	//���C�̕��������ݒ�
	char name[1000];
	//�����o���p�̖��O
	float *valuemap = new float[params.projection_div_phi*params.projection_div_theta];
	for (k = 0; k < params.projection_div_phi*params.projection_div_theta; k++) {
		valuemap[k] = 0;
	}
	//�]���l�̃�����

	printf("projection_setting_start\n");
	//////////////////////�ȉ�GPU�ł̃������m��
	float* d_proj1;
	float* d_ray_position;
	float* d_input_volume;
	//GPU�������m��
	hipMalloc(&d_proj1, sizeof(float)*params.projection_sides*params.projection_sides);
	hipMalloc(&d_ray_position, sizeof(float)*params.projection_sides*params.projection_sides * 3);
	hipMalloc(&d_input_volume, sizeof(float)*params.voxels_x*params.voxels_y*params.voxels_z);

	//�{�����[������GPU�ւƓ]��
	hipMemcpy(d_input_volume, input_volume, sizeof(float)*params.voxels_x*params.voxels_y*params.voxels_z, hipMemcpyHostToDevice);




	/////////////////////////�J��Ԃ��ΏہB���e�����A�]�����A�]���}�b�v���쐻////////////////////////////////////

	for (i = 0; i < params.projection_div_phi; i++) {//�J��Ԃ��񐔂̐ݒ�Bphi����,theta������]�œ�d���[�v�ɂȂ��Ă���B�B
		//theta������]���̏����ݒ�Bphi�ɏ������ړ�������B
		ray_theta = 0;
		ray_phi += PI / (params.projection_div_phi + 1);

		for (j = 0; j < params.projection_div_theta; j++) {

			printf("phi(%f)_theta(%f)\n",ray_phi, ray_theta);
			//////////////////////���e�������A���e�X�^�[�g�ʒu�ݒ�A���C�����ݒ�̏�����//////////////////////////

			for (k = 0; k < params.projection_sides*params.projection_sides; k++) {
				proj1[k] = 0;
			}//���e������������

			//���C�̕���������B�p�xtheta0phi0��(0,0,1)�B�������ɐL�тĂ��B�{�N�Z�����������̂܂܍��W�n�̒����ɑΉ����Ă��邽�߁A���̂܂܃X�e�b�v�Ƃ��ėp�����B
			ray_step[0] = sin(ray_phi)*cos(ray_theta)*STEP;
			ray_step[1] = sin(ray_phi)*sin(ray_theta)*STEP;
			ray_step[2] = cos(ray_phi)*STEP;
			printf("ray_x(%f) ray_y(%f) ray_z(%f)\n", ray_step[0], ray_step[1],ray_step[2]);

			ray_start_setting(params, ray_position, ray_phi, ray_theta);
			//�X�^�[�g�ʒu�ݒ�A�X�^�[�g�ʒu�ƁA�p�����[�^�𓊂�����ŉ�]������B



			//////////////////GPU�]������ьv�Z///////////////////////

			//GPU�ɓ��e�L�����o�X�Ɠ��e�ʒu�ݒ�f�[�^�]��
			hipMemcpy(d_proj1, proj1, sizeof(float)*params.projection_sides*params.projection_sides, hipMemcpyHostToDevice);
			hipMemcpy(d_ray_position, ray_position, sizeof(float)*params.projection_sides*params.projection_sides * 3, hipMemcpyHostToDevice);
			//hipMemcpy(d_input_volume, input_volume, sizeof(float)*params.voxels_x*params.voxels_y*params.voxels_z, hipMemcpyHostToDevice);

			printf("ray_marching_start\n");

			//�u���b�N���ݒ��GPU�v�Z�Ăяo��
			int BlockNum = (params.projection_sides*params.projection_sides + ThreadsPerBlock - 1) / ThreadsPerBlock;
			forward_marching_GPU << < BlockNum, ThreadsPerBlock >> > (d_input_volume, d_proj1, d_ray_position, ray_step[0], ray_step[1], ray_step[2], params);

			printf("ray_marching_end\n");
			
			//���e���̉���BCUDA�̌v�Z�����ׂďI����Ă���A���Ă���B
			hipMemcpy(proj1, d_proj1, sizeof(float)*params.projection_sides*params.projection_sides, hipMemcpyDeviceToHost);

	
			printf("memcpy_end\n");
	


			//////////////////////�]���l�̌v�Z////////////////////

			for (k = 0; k < params.projection_sides*params.projection_sides; k++) {
				valuemap[params.projection_div_theta*i + j] += proj1[k];
			}
			printf("value_%lf\n", valuemap[params.projection_div_theta*i + j]);
			



			//////////////////////�����o��(�I�v�V����)//////////////////////////
			if ( i % 20 == 0 && j % 10 == 0){
				printf("Writing projection\n");//���������͏����o���B�����o���t�@�C���Ɍ`���̎w��Ȃǂ͂Ȃ��B
				sprintf(name, "%s-float-%dx%d-(%f_%f).raw", params.out_name, params.projection_sides, params.projection_sides,ray_phi, ray_theta);
				printf("%s", name);
				FILE *out;
				out = fopen(name, "wb");
				if (out == NULL) {
					printf("\nFILE cannot open\n");
					exit(0);
				};
				fwrite(proj1, sizeof(float), params.projection_sides*params.projection_sides, out);
				fclose(out);
				printf("\nwriting_end\n\n");
			}
			//�R�}���h���C������̎��s�ł͏o�̓t�@�C���̓v���O�����̃t�H���_���ɂł���B���ڃe�L�X�g�𓊂����ނƃe�L�X�g�̂���t�H���_�ɂł���B
			



			//�p�x��theta�����ɍX�V
			ray_theta += PI / (params.projection_div_theta + 1);
		}
	}



	////////////////�������J��///////////////
	hipFree(d_input_volume);
	hipFree(d_proj1);
	hipFree(d_ray_position);	
	delete[] proj1;
	delete[] ray_position;	
	delete[] input_volume;
	////////////////�������J��///////////////

	//////////////////////�]���l�̔�r����////////////////////
	

	//�]���l�̔�r
	printf("Writing value map\n");
	sprintf(name, "valuemap-%s-float-(phi%d_theta%d).raw", params.out_name, params.projection_div_phi, params.projection_div_theta);
	printf("%s", name);
	FILE *out;
	out = fopen(name, "wb");
	if (out == NULL) {
		printf("\nFILE cannot open\n");
		exit(0);
	};
	fwrite(valuemap, sizeof(float), params.projection_div_phi*params.projection_div_theta, out);
	fclose(out);
	printf("\nwriting_end\n\n");
	//valuemap�̏����o��
	


	printf("\n projection_trajectory_analysis_start\n");
	float v_phi_y = 0, v_theta_x = 0;//value_map��ł̍��W
	int x, y;
	float axis_phi = 0, axis_theta = 0;//trajectory�̎��p�x�Btemp�Ɉꉞ�i�[���Ă���B
	float t = 0;//�O���̊p�x�B�J�X����B
	double trajectory_value = 0;
	double *trajectory_index = new double[(params.projection_div_phi - 1)*params.projection_div_theta];//�Ƃ肠�����A�]���O���͓��e���̐��ƈ�v������B

	for (i = 0; i < params.projection_div_phi - 1; i++) {
		axis_theta = 0;
		axis_phi += PI / params.projection_div_phi;
		for (j = 0; j < params.projection_div_theta; j++ ) {
			trajectory_value = 0;
			for (t = 0; t < PI; t += 0.05) {//����Ŕ���]
				v_phi_y = acos(-sin(axis_phi)*cos(t));
				v_theta_x = PI/2 + atan((-sin(axis_theta)*cos(axis_phi)*cos(t) + cos(axis_theta)*sin(t)) / (cos(axis_theta)*cos(axis_phi)*cos(t) - sin(axis_theta)*sin(t)));
				y = (int)(params.projection_div_phi - 1)*(v_phi_y / PI);
				x = (int)(params.projection_div_theta)*(v_theta_x / PI);
				trajectory_value += valuemap[y*params.projection_div_theta + x];
			}
			printf("trajectory value %lf phi %f theta %f\n", trajectory_value, axis_phi, axis_theta);
			trajectory_index[params.projection_div_theta*i + j] = trajectory_value;
			axis_theta += PI / params.projection_div_theta;
		}
	}//����Ŋe�O�����Ƃ̋P�x�l�̍��v�����[���ꂽ�B

	trajectory_value = trajectory_index[0];
	for (i = 0; i < params.projection_div_phi - 1; i++) {
		for (j = 0; j < params.projection_div_theta; j++) {
			if (trajectory_value > trajectory_index[params.projection_div_theta*i + j]) {
				trajectory_value = trajectory_index[params.projection_div_theta*i + j];
				x = j;
				y = i;
				axis_phi = x*PI / params.projection_div_phi;
				axis_theta = y*PI / params.projection_div_theta;
			}	
		}
	}
	printf("max_value_%lf axis phi_%f theta_%f\n", trajectory_value, axis_phi, axis_theta);


	////////////////�������J��///////////////
	delete[] trajectory_index;
	delete[] valuemap;
	
	////////////////�������J��///////////////

	printf("program_end\n");
	system("pause");
	return 0;
}