#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"

#include<stdio.h>
#include<string>
#include<fstream>
#include<math.h>

#define STEP 1
#define PI 3.1415926535897932384626433832795f //�~����
#define Filedivide 1�@//�ǂݍ��݃t�@�C���̕����B���ǎg��Ȃ������B
#define ThreadsPerBlock 512 //���݂� CUDA �ł� 512 �� MAX
#define angle_div 50 //�]�����̕�����

/*
���j

�c�Ɖ��̕������ɂ��ẮA�ő啪�����ݒ��A�i�ܓx���ƂɎ�����������B���������͕��������Ƃ͈ˑ����Ȃ��B
����Ă���͓̂��ߗ��̍��v�ł͂Ȃ��A���̍ő�l�݂̂𗘗p�B��蓧�ߑ�"�炵��"�Ȃ�

�]���l�̌v�Z�̈Ăɂ��Ă͈ȉ��̒ʂ�
�Ƃ肠�����A�摜�ꖇ�ɂ��P�x�����v�����]���l���Z�肵�A������e�p���ɑ΂��ĉ摜�]���l���Ƃɓ��ς��Ƃ��č��v�������̂����̎p���̍ŏI�]���l�Ƃ���B
�摜�ꖇ�̕]���l�́A�P�x�l�̍��v�Ƃ���B

�t�@�C���ǂݍ��ݏI�����_�ŁA���e�p�x�����Ǝ��s�p�x�񐔐ݒ���s���B
���e�L�����o�X�쐬�����C�}�[�`���O�v�Z���]���l�Z�o���������J���@�܂ł���A�̃v���Z�X�Ƃ��ă��W���[�����B
���e�o���t�@�C���Ɠ��e���t�@�C����main���ō쐬���āA��]�ݒ肾���O���֐��ɓ�����B

�t���O�����g������̂ŁAmalloc�Ƃ�free��for���̊O�B

*/

__global__ void forward_marching_GPU(unsigned short *d_input_volume, float *d_proj1, float *d_ray_position, float step_x, float step_y, float step_z, Params params) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	int j;
	float v = 0;//�P�x�̊i�[��
	d_proj1[idx] = 0;
	//���C���ɂ��ēƗ��v�Z���Ă��邽�߁Aid�͈�ł����B
		for (j = 0; j < params.projection_radius / 2.0 / STEP; j++) {//���v���X�e�b�v�𓥂ނ���j�ŕ\���Ă���
			int x = (int)(d_ray_position[idx * 3] + j*step_x);//�����̈�X�e�b�v���ƂɁAxyz�����ɉ��{�N�Z�����i�ނ����v�Z���Aint�Ŋۂ߂��čŏI�I�ȓ_���W�𓾂Ă���B
			int y = (int)(d_ray_position[idx * 3 + 1] + j*step_y);
			int z = (int)(d_ray_position[idx * 3 + 2] + j*step_z);
			if (x > 0 && y > 0 && z > 0 && x < params.voxels_x && y < params.voxels_y && z < params.voxels_z) {
				//������̓_���A�{�����[���̒u���Ă���{�b�N�X��Ԃɂ��鎞�A
				//v += d_input_volume[x + y * params.voxels_x + z * params.voxels_x * params.voxels_y];//���ɂ��̍��\��ɂ���g���O���t�̃{�N�Z���̒l�����Z���Ă����B
				if (v < d_input_volume[x + y * params.voxels_x + z * params.voxels_x * params.voxels_y]) {//�ő�l���Ƃ�B
					v = d_input_volume[x + y * params.voxels_x + z * params.voxels_x * params.voxels_y];
				}
			}
			int x2 = (int)(d_ray_position[idx * 3] - j*step_x);//�t�����ɂ��}�[�`���O����B
			int y2 = (int)(d_ray_position[idx * 3 + 1] - j*step_y);
			int z2 = (int)(d_ray_position[idx * 3 + 2] - j*step_z);
			if (x2 > 0 && y2 > 0 && z2 > 0 && x2 < params.voxels_x && y2 < params.voxels_y && z2 < params.voxels_z) {
				//������̓_���A�{�����[���̒u���Ă���{�b�N�X��Ԃɂ��鎞�A
				//v += d_input_volume[x2 + y2 * params.voxels_x + z2 * params.voxels_x * params.voxels_y];//���ɂ��̍��\�ɂ���g���O���t�̃{�N�Z���̒l�����Z���Ă����B
				if (v < d_input_volume[x2 + y2 * params.voxels_x + z2 * params.voxels_x * params.voxels_y]) {//�ő�l���Ƃ�B
					v = d_input_volume[x2 + y2 * params.voxels_x + z2 * params.voxels_x * params.voxels_y];
				}
			}
		}
		d_proj1[idx] = v;
}


void ray_start_setting(Params params, float *ray_position, float ray_phi, float ray_theta) {
	int i, j;
	//�������B���W���S��ɂ����ă��C�̃X�^�[�g�ʒu��ݒ�Bi���L�����o�X��ł�y���W,j���L�����o�X��ł�x���W�ɑΉ��B
	for (i = 0; i < params.projection_sides; i++) {
		for (j = 0; j < params.projection_sides; j++) {
			ray_position[(i*params.projection_sides + j) * 3] = j - params.projection_radius / 2;
			ray_position[(i*params.projection_sides + j) * 3 + 1] = i - params.projection_radius / 2;
			ray_position[(i*params.projection_sides + j) * 3 + 2] = 0;
		}
	}

	//�X�^�[�g���W��y,z���ɑ΂���theta,phi������]�������̂��A���s�ړ�������
	float a, b, c;
	for (i = 0; i < params.projection_sides*params.projection_sides; i++) {
		//�܂�y���ɂ����ĉ�]
		a = ray_position[i * 3];
		b = ray_position[i * 3 + 1];
		c = ray_position[i * 3 + 2];
		ray_position[i * 3] = cos(ray_phi)*a + sin(ray_phi)*c;
		ray_position[i * 3 + 1] = b;
		ray_position[i * 3 + 2] = -sin(ray_phi)*a + cos(ray_phi)*c;
	 //����z���ɂ����ĉ�]
		a = ray_position[i * 3];
		b = ray_position[i * 3 + 1];
		c = ray_position[i * 3 + 2];
		ray_position[i * 3] = cos(ray_theta)*a - sin(ray_theta)*b;
		ray_position[i * 3 + 1] = sin(ray_theta)*a + cos(ray_theta)*b;
		ray_position[i * 3 + 2] = c;
		//�{�����[���̒��S���W�ɕ��s�ړ�

		ray_position[i * 3] = ray_position[i * 3] + params.voxels_x / 2.0 - 0.5;
		ray_position[i * 3 + 1] = ray_position[i * 3 + 1] + params.voxels_y / 2.0 - 0.5;
		ray_position[i * 3 + 2] = ray_position[i * 3 + 2] + params.voxels_z / 2.0 - 0.5;
		//printf("ray_2_position(%f,%f,%f)\n", ray_position[i * 3], ray_position[i * 3 + 1], ray_position[i * 3 + 1]);
	}

}



int main(int argc, char** argv) {//argc�Ƃ��ɂ͋N�����ɓn���ϐ����͂���B

	int i, j, k, l;//�֗��ɃJ�E���g�ȂǂɎg������
	Params params;//�p�����[�^�\����
	FILE *in, *para;//���̓t�@�C���p�̃t�@�C���e��
						  //���̓t�@�C���Ɠ����ϐ��Ԃ̐����̂����̈ꎞ�I�ȃo�b�t�@

	//////////////////////�p�����[�^�ǂݍ���//////////////////////////

	if ((para = fopen(argv[1], "r")) == NULL) {
		printf("FILE do not exist\n");
		exit(0);
	};
	
	//�ݒ�t�@�C������p�����[�^�ǂݍ��݁B�X�y�[�X���󂯂�Ǝ��ɍs���̂Ńt�@�C�����ɃX�y�[�X�͂��Ȃ��悤�ɁB
	fscanf(para, "%s", params.in_path_name);
	fscanf(para, "%s", params.out_name);
	fscanf(para, "%d", &params.in_offset);
	fscanf(para, "%f %f", &params.source_object_distance, &params.source_detector_distance);//�����g��Ȃ�
	fscanf(para, "%d %d %d", &params.voxels_x, &params.voxels_y, &params.voxels_z);
	fscanf(para, "%f %f", &params.projection_phi, &params.projection_theta);//�����̏����ݒ�B���܂����Ȃ�
	fclose(para);

	params.projection_div_phi = 4;//�Ƃ肠������`�B������scan�ł�݂��܂��Ă��悢�B
	params.projection_div_theta = 4;

	printf("projection angle division number in form of phi theta\n");//���e�p�x�̖����̐ݒ�B�����œǂݍ��ޓ��e�͂悭�ύX����B
	scanf_s("%d %d", &params.projection_div_phi, &params.projection_div_theta);

	params.projection_radius = sqrt(params.voxels_x * params.voxels_x + params.voxels_y *params.voxels_y + params.voxels_z * params.voxels_z);//���e���̈�Ђ̒���
	params.projection_sides = (int)params.projection_radius;//�{�N�Z�������A�܂萮���Ɋۂ߂��ꍇ�B

	printf(" input %s\n output %s\n distance %f %f\n voxels %d %d %d\n start angle phi %f theta %f\n angle division phi %d theta %d\n projection radius %f projection sides %d",
		params.in_path_name, params.out_name,
		params.source_object_distance, params.source_detector_distance,
		params.voxels_x, params.voxels_y, params.voxels_z,
		params.projection_phi, params.projection_theta,
		params.projection_div_phi, params.projection_div_theta,
		params.projection_radius,params.projection_sides);//�p�����[�^�����o���B
	//////////////////////�p�����[�^�ǂݍ��ݏI��//////////////////////////

	//////////////////////�ǂݍ��݂Ɗe�탁�����m��//////////////////////////
	if ((in = fopen(params.in_path_name, "rb")) == NULL) {
		printf("FILE do not exist_1\n");
		exit(0);//�R�}���h���C������̎��s�ł̓t���̃p�X���w�肵�Ȃ���Ό�����Ȃ��B������\��2�񂩂��B���邢�͐ݒ�e�L�X�g�𒼐ڃv���O�����ɕ��Ă��悢�B
	}
	printf("load_success\n");
	
	unsigned short *input_volume = new unsigned short[params.voxels_y*params.voxels_x*params.voxels_z];
	//���ʃT�C�Y�~���݂�1�����z��쐬
	printf("memory_success\n");

	fseek(in, params.in_offset, SEEK_SET);//�ǂݍ��ݎ��ɃI�t�Z�b�g�ړ�
	fread(input_volume, 2, params.voxels_x*params.voxels_y*params.voxels_z, in);//�{�����[���ǂݍ���
	fclose(in);
	printf("loading_success\n");
	
	float *proj1 = new float[params.projection_sides*params.projection_sides];
	//���e���̃������m��
	float *ray_position = new float[params.projection_sides*params.projection_sides * 3];
	//�L�����o�X�T�C�Y��3�{�̓��e���C�̃X�^�[�g�ʒu���i�[���郁�����Bxyz���W�Őݒ�
	float ray_step[3];
	//���C�̊i�[�e��쐬
	float ray_phi = params.projection_phi;
	float ray_theta = params.projection_theta;
	//���C�̕���
	char name[1000];
	//�����o���p�̖��O
	float *temp = new float[params.projection_div_phi*params.projection_div_theta*3];
	float *valuemap = new float[params.projection_div_phi*params.projection_div_theta];
	//�]���l�Ɠ��e�p�x�̃�����

	printf("projection_setting_start\n");
	//////////////////////�ȉ�GPU�ł̃������m��
	float* d_proj1;
	float* d_ray_position;
	unsigned short* d_input_volume;
	//GPU�������m��
	hipMalloc(&d_proj1, sizeof(float)*params.projection_sides*params.projection_sides);
	hipMalloc(&d_ray_position,sizeof(float)*params.projection_sides*params.projection_sides * 3);
	hipMalloc(&d_input_volume, sizeof(unsigned short)*params.voxels_x*params.voxels_y*params.voxels_z);

	/////////////////////�ǂݍ��݂Ɗe�탁�����m�ۏI��//////////////////////////


	for (i = 0; i < params.projection_div_phi*params.projection_div_theta; i++) {//�J��Ԃ��񐔂̐ݒ�Bphi����,theta������]�œ�d���[�v�ɂȂ��Ă���B�B

		//////////////////////���e�������A���e�X�^�[�g�ʒu�ݒ�A���C�����ݒ�̏�����//////////////////////////

		for (k = 0; k < params.projection_sides*params.projection_sides; k++) {
			proj1[k] = 0;
		}//���e������������

		ray_step[0] = sin(ray_phi)*cos(ray_theta)*STEP;
		ray_step[1] = sin(ray_phi)*sin(ray_theta)*STEP;
		ray_step[2] = cos(ray_phi)*STEP;
		//���C�̕���������B�p�xtheta0phi0��(0,0,1)�B�������ɐL�тĂ��B�{�N�Z�����������̂܂܍��W�n�̒����ɑΉ����Ă��邽�߁A���̂܂܃X�e�b�v�Ƃ��ėp�����B
	
		ray_start_setting(params, ray_position,ray_phi,ray_theta);
		//�X�^�[�g�ʒu�ݒ�A�X�^�[�g�ʒu�ƁA�p�����[�^�𓊂�����ŉ�]������B

		//printf("ray,ray_canvas,ray_start_positions are successfully_created\n");
		//printf("ray,%f,%f,%f\n",ray_step[0],ray_step[1],ray_step[2]);

		//////////////////////���e�������I��//////////////////////////


		//////////////////GPU�]������ьv�Z///////////////////////

		//GPU�Ƀf�[�^�]��
		hipMemcpy(d_proj1, proj1, sizeof(float)*params.projection_sides*params.projection_sides, hipMemcpyHostToDevice);
		hipMemcpy(d_ray_position, ray_position, sizeof(float)*params.projection_sides*params.projection_sides * 3, hipMemcpyHostToDevice);
		hipMemcpy(d_input_volume, input_volume, sizeof(unsigned short)*params.voxels_x*params.voxels_y*params.voxels_z, hipMemcpyHostToDevice);
	
		printf("ray_marching_start\n");
	
		//�u���b�N���ݒ��GPU�v�Z�Ăяo��
		int BlockNum = (params.projection_sides*params.projection_sides + ThreadsPerBlock - 1) / ThreadsPerBlock;
		forward_marching_GPU <<< BlockNum, ThreadsPerBlock >>> ( d_input_volume,  d_proj1, d_ray_position, ray_step[0], ray_step[1], ray_step[2], params);

		//���e���̉��
		hipMemcpy(proj1, d_proj1, sizeof(float)*params.projection_sides*params.projection_sides, hipMemcpyDeviceToHost);
	
		printf("ray_marching_end\n");

		//////////////////////GPU�]���v�Z�����//////////////////////////


		//////////////////////�]���l�̌v�Z////////////////////�p�x���ƂɋP�x��ώZ���Ă���l�B���̃f�[�^�̈������͏����ς������B�����摜�ŕ\����悤�ɂ������B
		temp[3*i+1] = params.projection_phi;
		temp[3*i+2] = params.projection_theta;
		for (j = 0; j < params.projection_sides*params.projection_sides; j++) {
			temp[3*i] += proj1[j];
			valuemap[i] += proj1[j];//���ꂪ�e�p�x�̕]���l�����[����摜�ɂȂ�B
		}
		//////////////////////�]���l�̌v�Z�I��/////////////////////////


		//////////////////////�����o��(�I�v�V����)//////////////////////////
		/*
		printf("picture pixel %f\n",proj1[params.projection_sides*params.projection_sides/4]);
		printf("Writing\n");//���������͏����o���B�����o���t�@�C���Ɍ`���̎w��Ȃǂ͂Ȃ��B
		sprintf(name, "%s-float-%dx%d-(%f_%f).raw", params.out_name, params.projection_sides, params.projection_sides,params.projection_phi, params.projection_theta);
		printf("%s", name);
		FILE *out;
		out = fopen(name, "wb");
		if (out == NULL) {
			printf("\nFILE cannot open\n");
			exit(0);
		};
		//out�ɓ��e�L�����o�X���Ԃ����ށB�Ƃ肠�����̓t���[�g��
		fwrite(proj1, sizeof(float), params.projection_sides*params.projection_sides, out);
		fclose(out);
		printf("\nwriting_end\n\n");

		//�R�}���h���C������̎��s�ł͏o�̓t�@�C���̓v���O�����̃t�H���_���ɂł���B���ڃe�L�X�g�𓊂����ނƃe�L�X�g�̂���t�H���_�ɂł���B
		*/
		//////////////////////�����o���I��//////////////////////////


		//////////////////////�J��Ԃ��̎��̎p���X�V//////////////////////////
		if ( ( i + 1 ) % params.projection_div_theta == 0 ) {
			ray_phi = ray_phi + PI / params.projection_div_phi / 2;
			ray_theta = 0;
		}
		else {
			ray_theta = ray_theta + PI / params.projection_div_theta / 2;
		}
		//////////////////////�p���X�V�I��//////////////////////////

	}



	////////////////�������J��///////////////
	hipFree(d_input_volume);
	hipFree(d_proj1);
	hipFree(d_ray_position);	
	delete[] proj1;
	delete[] ray_position;	
	delete[] input_volume;
	////////////////�������J��///////////////

	//////////////////////�]���l�̔�r����////////////////////
	/**/
	float vphi, vtheta;
	float *value = new float[angle_div*angle_div];
	float max_value[3];
	for (i = 0; i < params.projection_div_theta*params.projection_div_phi; i++) {
		printf("value %f (phi %f theta %f)\n", temp[3*i], temp[3*i + 1], temp[3*i + 2]);

		for (j = 0; j < angle_div; j++) {
			for (k = 0; k < angle_div; k++) {
				value[j*angle_div + k] = 0;
			}
		}
		for (j = 0; j < angle_div; j++) {
			vphi = PI*2.0f*j/angle_div;
			for (k = 0; k < angle_div; k++) {
				vtheta = PI*2.0f*k / angle_div;
				for (l = 0; l < params.projection_div_phi * params.projection_div_theta ; l++) {
					value[j*angle_div + k] += temp[3*i]*(sin(temp[3 * l + 1])*cos(temp[3 * l + 2])*sin(vphi)*cos(vtheta) + sin(temp[3 * l + 1])*sin(temp[3 * l + 2])*sin(vphi)*sin(vtheta) + cos(temp[3 * l + 1])*cos(vphi));
				}
				//printf("value %f phi %f theta %f\n", value[j*angle_div + k], vphi, vtheta);
				if (max_value[0]<value[j*angle_div + k]) {
					max_value[0] = value[j*angle_div + k];
					max_value[1] = vphi;
					max_value[2] = vtheta;
				}
			}
		}
	}

	printf("max_value %f phi %f theta %f\n", max_value[0], max_value[1], max_value[2]);
	for (i = 0; i < params.projection_div_phi*params.projection_div_theta;i++) {
		printf("value %f \n", valuemap[i]);

	}
	//////////////////////�]���l�̔�r�����I��////////////////

	///�]���l�̔�r��ځB
	printf("Writing value map\n");
	sprintf(name, "valuemap-%s-float-(%d_%d).raw", params.out_name, params.projection_div_theta, params.projection_div_phi);
	printf("%s", name);
	FILE *out;
	out = fopen(name, "wb");
	if (out == NULL) {
		printf("\nFILE cannot open\n");
		exit(0);
	};
	fwrite(valuemap , sizeof(float), params.projection_div_phi*params.projection_div_theta, out);
	fclose(out);
	printf("\nwriting_end\n\n");
	double v_a_phi = 0, v_a_theta = 0;
	double v_phi = 0, v_theta = 0;
	double t = 0;
	for (t = 0; t < PI; t += 0.01) {//����ň��]
		v_a_phi = acos(-sin(v_a_phi)*cos(t));
		v_a_theta = atan((-sin(v_a_phi)*cos(v_a_theta)*cos(t) + cos(v_a_phi)*sin(t)) / (cos(v_a_phi)*cos(v_a_theta)*cos(t) - sin(v_a_phi)*sin(t)));
	}



	////////////////�������J��///////////////
	delete[] valuemap;
	delete[] temp;
	delete[] value;
	////////////////�������J��///////////////

	printf("program_end\n");

	return 0;
}