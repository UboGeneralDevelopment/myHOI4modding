
#include "hip/hip_runtime.h"


#include<stdio.h>
#include<string>
#include<fstream>
#include<math.h>


int main(int argc, char** argv) {
	int i, j, k, l;
	FILE *volumef1, *reliabilityf1, *volumef2, *reliabilityf2, *para;

	//�{�����[���A�M���x�A��]�s��Ƃ����̂ق��p�����[�^

	/*
	�ݒ�t�@�C���p�����[�^

	1�t�@�C��1�̃p�X
	1�t�@�C��2�̃p�X
	�o�̓t�@�C����
	�{�����[���̃{�N�Z����XYZ�A

	*/

	char in_path_1[1000], in_path_2[1000], out_name[1000];

	//���̓t�@�C���Əo�̓t�@�C����
	int voxels[3];
	//double vsize;

	if ((para = fopen(argv[1], "r")) == NULL) {
		printf("FILE do not exist\n");
		exit(0);
	};

	fscanf(para, "%s", in_path_1);
	fscanf(para, "%s", in_path_2);
	fscanf(para, "%s", out_name);
	fscanf(para, "%d %d %d", &voxels[0], &voxels[1], &voxels[2]);
	fclose(para);

	//�{�����[���̊i�[�e��@xy���ʃT�C�Yvoxels1[0] * voxels1[1]�@x���T�C�Yvoxel1[0]
	float *volume_1 = new float[voxels[0] * voxels[1] * voxels[2]];

	float *volume_2 = new float[voxels[0] * voxels[1] * voxels[2]];

	float *fusioned = new float[voxels[0] * voxels[1] * voxels[2]];

	//�{�����[���ǂݍ���
	if ((volumef1 = fopen(in_path_1, "rb")) == NULL) {
		printf("FILE do not exist_1\n");
		exit(0);
	}
	printf("volume 1 load success\n");
	fread(volume_1, sizeof(float), voxels[0] * voxels[1] * voxels[2], volumef1);
	fclose(volumef1);


	if ((volumef2 = fopen(in_path_2, "rb")) == NULL) {
		printf("FILE do not exist_1\n");
		exit(0);
	}
	printf("volume 2 load success\n");
	fread(volume_2, sizeof(float), voxels[0] * voxels[1] * voxels[2], volumef2);//�{�����[���ǂݍ���
	fclose(volumef2);

	printf("fusion start\n");

	float r1, r2;

	for (i = 0; i < voxels[2]; i++) {
		printf("progress %d / %d\n", i, voxels[2]);
		for (j = 0; j < voxels[1]; j++) {
			for (k = 0; k < voxels[0]; k++) {
				/*
				�����ŐM���x�����v�Z
				*/
				r1 = volume_1[voxels[0] * voxels[1] * i + voxels[0] * j + k];
				r2 = volume_2[voxels[0] * voxels[1] * i + voxels[0] * j + k] + 0.0005;

				fusioned[voxels[0] * voxels[1] * i + voxels[0] * j + k] = r1*r2;
			}
		}
	}

	printf("Writing fusioned volume\n");
	char name[1000];
	sprintf(name, "%s-float-(%dx%dx%d).raw", out_name, voxels[0], voxels[1], voxels[2]);
	printf("%s", name);

	FILE *out;
	out = fopen(name, "wb");
	if (out == NULL) {
		printf("\nFILE cannot open\n");
		exit(0);
	};
	fwrite(fusioned, sizeof(float), voxels[0] * voxels[1] * voxels[2], out);

	printf("\nWriting fusioned volume end\n");

	fclose(out);


	delete[] volume_1;
	delete[] volume_1;
	delete[] fusioned;
	return 0;
}