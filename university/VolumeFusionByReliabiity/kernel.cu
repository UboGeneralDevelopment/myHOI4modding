
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <string>

int main(int argc, char** argv){
	int i, j, k, l;
	FILE *volumef1, *reliabilityf1, *volumef2, *reliabilityf2, *para;

	//ボリューム、信頼度、回転行列とかそのほかパラメータ
	
	/*
	設定ファイルパラメータ

	1ボリュームファイルのパス
	1信頼度ファイルのパス
	2ボリュームファイルのパす
	2信頼度ファイルのパス
	出力ファイル名
	ボリュームのボクセル数XYZ、

	*/

	char in_path_v1[1000], in_path_r1[1000], in_path_v2[1000], in_path_r2[1000], out_name[1000];
	
	//入力ファイルと出力ファイル名
	int voxels[3];
	//double vsize;

	if ((para = fopen(argv[1], "r")) == NULL) {
		printf("FILE do not exist\n");
		exit(0);
	};

	fscanf(para, "%s", in_path_v1);
	fscanf(para, "%s", in_path_r1);
	fscanf(para, "%s", in_path_v2);
	fscanf(para, "%s", in_path_r2);
	fscanf(para, "%s", out_name);
	fscanf(para, "%d %d %d", &voxels[0], &voxels[1], &voxels[2]);
	fclose(para);

	//ボリュームの格納容器　xy平面サイズvoxels1[0] * voxels1[1]　x一列サイズvoxel1[0]
	float *volume_1 = new float[voxels[0] * voxels[1] * voxels[2]];

	float *reliability_1 = new float[voxels[0] * voxels[1] * voxels[2]];
	
	float *volume_2 = new float[voxels[0] * voxels[1] * voxels[2]];

	float *reliability_2 = new float[voxels[0] * voxels[1] * voxels[2]];

	float *fusioned = new float[voxels[0] * voxels[1] * voxels[2]];

	float *fusioned_reliability = new float[voxels[0] * voxels[1] * voxels[2]];

	//ボリューム読み込む
	if ((volumef1 = fopen(in_path_v1, "rb")) == NULL) {
		printf("FILE do not exist_1\n");
		exit(0);
	}
	printf("volume 1 load success\n");
	fread(volume_1, sizeof(float), voxels[0] * voxels[1] * voxels[2], volumef1);
	fclose(volumef1);


	if ((volumef2 = fopen(in_path_v2, "rb")) == NULL) {
		printf("FILE do not exist_1\n");
		exit(0);
	}
	printf("volume 2 load success\n");
	fread(volume_2, sizeof(float), voxels[0] * voxels[1] * voxels[2], volumef2);//ボリューム読み込み
	fclose(volumef2);


	if ((reliabilityf1 = fopen(in_path_r1, "rb")) == NULL) {
		printf("FILE do not exist_1\n");
		exit(0);
	}
	printf("reliability 1 load success\n");
	fread(reliability_1, sizeof(float), voxels[0] * voxels[1] * voxels[2], reliabilityf1);//ボリューム読み込み
	fclose(reliabilityf1);


	if ((reliabilityf2 = fopen(in_path_r2, "rb")) == NULL) {
		printf("FILE do not exist_1\n");
		exit(0);
	}
	printf("reliability 2 load success\n");
	fread(reliability_2, sizeof(float), voxels[0] * voxels[1] * voxels[2], reliabilityf2);//ボリューム読み込み
	fclose(reliabilityf2);


	printf("fusion start\n");

	float r1, r2, tr;
	float v1, v2, vf, vr;
	for (i = 0; i < voxels[2]; i++) {
		printf("progress %d / %d\n", i, voxels[2]);
		for (j = 0; j < voxels[1]; j++) {
			for (k = 0; k < voxels[0]; k++) {

				r1 = reliability_1[voxels[0] * voxels[1] * i + voxels[0] * j + k];
				r2 = reliability_2[voxels[0] * voxels[1] * i + voxels[0] * j + k];

				vr = (r1*r2) / (r1 + r2);

				fusioned_reliability[voxels[0] * voxels[1] * i + voxels[0] * j + k] = vr;

				r1 = 1 / r1;
				r2 = 1 / r2;
				tr = r1 + r2;
				r1 = r1 / tr;
				r2 = r2 / tr;

				v1 = volume_1[voxels[0] * voxels[1] * i + voxels[0] * j + k];
				v2 = volume_2[voxels[0] * voxels[1] * i + voxels[0] * j + k];

				vf = v1*r1 + v2*r2;

				fusioned[voxels[0] * voxels[1] * i + voxels[0] * j + k] = vf;

			}
		}
	}
	char name[1000];
	FILE *out;

	printf("Writing fusioned volume\n");

	sprintf(name, "%s-float-(%dx%dx%d).raw", out_name, voxels[0], voxels[1], voxels[2]);
	printf("%s", name);


	out = fopen(name, "wb");
	if (out == NULL) {
		printf("\nFILE cannot open\n");
		exit(0);
	};
	fwrite(fusioned, sizeof(float), voxels[0] * voxels[1] * voxels[2], out);

	printf("\nWriting fusioned volume end\n");

	fclose(out);

	printf("Writing fusioned reliability\n");

	sprintf(name, "%s-reliability-float-(%dx%dx%d).raw", out_name, voxels[0], voxels[1], voxels[2]);
	printf("%s", name);


	out = fopen(name, "wb");
	if (out == NULL) {
		printf("\nFILE cannot open\n");
		exit(0);
	};
	fwrite(fusioned_reliability, sizeof(float), voxels[0] * voxels[1] * voxels[2], out);

	printf("\nWriting fusioned reliability end\n");

	fclose(out);

	/*
	printf("bit adusted ver writing\n");

	for (i = 0; i < voxels[2]; i++) {
		printf("progress %d / %d\n", i, voxels[2]);
		for (j = 0; j < voxels[1]; j++) {
			for (k = 0; k < voxels[0]; k++) {


				fusioned[voxels[0] * voxels[1] * i + voxels[0] * j + k] = 1000*fusioned[voxels[0] * voxels[1] * i + voxels[0] * j + k];
			}
		}
	}
	sprintf(name, "%s-float-adjusted-(%dx%dx%d).raw", out_name, voxels[0], voxels[1], voxels[2]);
	out = fopen(name, "wb");
	if (out == NULL) {
		printf("\nFILE cannot open\n");
		exit(0);
	};
	fwrite(fusioned, sizeof(float), voxels[0] * voxels[1] * voxels[2], out);

	printf("\nWriting fusioned volume end\n");

	fclose(out);
*/

	delete[] volume_1;
	delete[] volume_1;
	delete[] reliability_1;
	delete[] reliability_2;
	delete[] fusioned;
	delete[] fusioned_reliability;
    return 0;
}
